#include "hip/hip_runtime.h"
﻿//Carlos Miguel Negrete
//A01208733

//Programming Languajes Final Project

//Dependencies:
//	- Opencv2 --Version 4.5.2
//	- Some images for testing. I provide a bounch of  images for usage.
//I used:
//	- Nvidia GeForcce GTX 960m compute_50,sm_50

#include <iostream>
#include <stdio.h>
#include "string"
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include "hip/hip_runtime.h"
#include ""
#include <hip/device_functions.h>

#define N 32 //Threads per block

using namespace std;
using namespace cv;

__global__ void filter1(uint8_t* image, int width, int height, int Channels) {
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int ind = (x + y * gridDim.x * blockDim.x) * Channels;

	//Changes the color  from "Black" to something kind of red
	if ((image[ind + 0] <= 70) && (image[ind + 1] <= 70) && (image[ind + 2] <= 70)) {
		image[ind + 2] = image[ind + 2] + 80;
	}
	
}

//Adds vertical black lines to the image, if it is PNG it adds a different pattern since the number of channels. 
__global__ void filter2(uint8_t* image, int width, int height, int Channels) {
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int ind = (x + y * gridDim.x * blockDim.x) * Channels;

	if (Channels < 4) {
		if ((ind % 2) == 0) {
			for (int i = 0; i < Channels; i++) {
				image[ind + i] = 0;
			}
		}
	}
	else {
		if ((ind % 3) == 0) {
			for (int i = 0; i < Channels; i++) {
				image[ind + i] = 0;
			}
		}
	}
	
}

//Set tranparenecy from white color. 
__global__ void filter3(uint8_t* image, int width, int height, int Channels) {
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int ind = (x + y * gridDim.x * blockDim.x) * Channels;

	if ((image[ind + 0] >= 91) && (image[ind + 1] >= 91) && (image[ind + 2] >= 91)) {
		image[ind + 3] = 0;
	}

}

int main() {
	//List actual directory
	system("dir"); //For Windows users
	//system("ls"); //Uncomment this for Unix based SO's

	//Timestamp purposes
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);



	//Instructions for the user
	cout << endl;
	cout << "------------- Welcome to my Image Color and More changer program.-------------" << endl << "- Please write the name of the image you want to transform" << endl << "- (include filename extension), you have your actual directory above. -" << endl;
	
	//ASking for the original Image
	//string imageName;

	//cin >> imageName;

	//Variables creation
	//Mat image = imread(imageName, IMREAD_UNCHANGED);

	string imageName = "img4v3.jpg";
	cout << imageName << endl;
	Mat image = imread(imageName, IMREAD_UNCHANGED);

	uint8_t* d_image, *d_image2, *d_image3;
	int pixelNumValues = image.step;
	int h = image.rows;
	int w = image.cols;

	//Displaying some windows for preview the images.
	namedWindow(imageName + " ORIGINAL", WINDOW_NORMAL);
	resizeWindow(imageName + " ORIGINAL", 400, 400);
	imshow(imageName + " ORIGINAL", image);

	//Creating the blocks and the threads.
	dim3 blocks(h / N, w / N);
	dim3 threads(N, N);
	
	//Memory allocation in Device and copy of info for filter 1
	hipMalloc((void**)&d_image, sizeof(uint8_t) * (pixelNumValues * h));
	hipMemcpy(d_image, image.data, sizeof(uint8_t) * (pixelNumValues * h), hipMemcpyHostToDevice);

	//Memory allocation in Device and copy of info for filter 2
	hipMalloc((uint8_t**)&d_image2, sizeof(uint8_t) * (pixelNumValues * h));
	hipMemcpy(d_image2, image.data, sizeof(uint8_t) * (pixelNumValues * h), hipMemcpyHostToDevice);

	//Memory allocation in Device and copy of info for filter 3 only if it is a PNG image
	if (image.channels() == 4) {
		hipMalloc((uint8_t**)&d_image3, sizeof(uint8_t) * (pixelNumValues * h));
		hipMemcpy(d_image3, image.data, sizeof(uint8_t) * (pixelNumValues * h), hipMemcpyHostToDevice);
	}
	

	//Used only to show the number of channels (4 if it has tranparency capacity or 3 if normal image)
	//cout << image.channels();

	//filter1 changes blackish colours to redish tones. 
	hipEventRecord(start, 0);
	filter1 << <blocks, threads >> > (d_image, w, h, image.channels());
	hipEventRecord(stop, 0);
	hipDeviceSynchronize();
	hipMemcpy(image.data, d_image, sizeof(uint8_t) * (pixelNumValues * h), hipMemcpyDeviceToHost);
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	cout << "Time elapsed for filter 1: " << milliseconds << " ms" << endl;
	hipFree(d_image);

	namedWindow(imageName + "RED", WINDOW_NORMAL);
	resizeWindow(imageName + "RED", 400, 400);
	imshow(imageName + "RED", image);
	imwrite(imageName + "RED.png", image);

	//Filter 2 adds black pattern
	hipEventRecord(start, 0);
	filter2 << <blocks, threads >> > (d_image2, w, h, image.channels());
	hipEventRecord(stop, 0);
	hipDeviceSynchronize();
	hipMemcpy(image.data, d_image2, sizeof(uint8_t) * (pixelNumValues * h), hipMemcpyDeviceToHost);
	hipEventSynchronize(stop);
	float milliseconds2 = 0;
	hipEventElapsedTime(&milliseconds2, start, stop);
	cout << "Time elapsed for filter 2: " << milliseconds2 << " ms" << endl;
	hipFree(d_image2);

	namedWindow(imageName + "LINES", WINDOW_NORMAL);
	resizeWindow(imageName + "LINES", 400, 400);
	imshow(imageName + "LINES", image);
	imwrite(imageName + "LINES.png", image);

	//Filter 3 only if image is a png
	if (image.channels() == 4) {
		hipEventRecord(start, 0);
		filter3 << <blocks, threads >> > (d_image3, w, h, image.channels());
		hipEventRecord(stop, 0);
		hipDeviceSynchronize();
		hipMemcpy(image.data, d_image3, sizeof(uint8_t) * (pixelNumValues * h), hipMemcpyDeviceToHost);
		hipEventSynchronize(stop);
		float milliseconds3 = 0;
		hipEventElapsedTime(&milliseconds3, start, stop);
		cout << "Time elapsed for filter 3: " << milliseconds3 << " ms" << endl;
		hipFree(d_image3);

		namedWindow(imageName + "TRANPARENCY", WINDOW_NORMAL);
		resizeWindow(imageName + "TRANPARENCY", 400, 400);
		imshow(imageName + "TRANPARENCY", image);
		imwrite(imageName + "TRANPARENCY.png", image);
	}

	waitKey(0);

	return 0;
}